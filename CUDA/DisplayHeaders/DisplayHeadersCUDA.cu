//Display some infor about the card and machine


//#include <cuda.h>


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

//http://stackoverflow.com/questions/5689028/how-to-get-card-specs-programatically-in-cuda/5689133#5689133
void CUDA_Info()
{
using namespace std;
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << endl << "NBody.GPU" << endl << "=========" << endl << endl;

#ifdef CUDART_VERSION
    wcout << "CUDA version:   v" << CUDART_VERSION << endl;  
#endif  

#ifdef THRUST_MAJOR_VERSION
    wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 
#endif

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}
